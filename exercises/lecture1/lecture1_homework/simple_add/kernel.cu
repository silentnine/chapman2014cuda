#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    /* sum of a and b stored in c */
	...
}

int main()
{
    int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* setup initial values */

	a = 2;
	b = 7;
	c = -99;

	/* copy inputs to device */

	hipMemcpy( d_a, &a, size, ... );
	hipMemcpy( d_b, &b, size, ... );

	/* launch the kernel on the GPU */
	/* use 1 block per grid and 1 thread per block*/
	...

	/* copy result back to host */

	hipMemcpy( &c, d_c, size, ... );

	printf("value of c after kernel is %d\n",c);

	/* clean up */

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
