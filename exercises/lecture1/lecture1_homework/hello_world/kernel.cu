#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void mykernel(){
	printf("Hello world from device!\n");
} /* end kernel */

int main(void) 
{
        /* launch this kernel 10 times*/
	mykernel<<< ... , ...>>>();
 	hipDeviceSynchronize();
	printf("Hello World from Host\n");
	return 0;
} /* end main */
